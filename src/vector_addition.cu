#include "hip/hip_runtime.h"
#include "../include/test.cuh"

// CUDA kernel to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void test(int* c_i, int N)
{
   const int n = 1000; // Size of the vectors
    int *a, *b, *c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory on the host
    a = new int[n];
    b = new int[n];

    // Initialize vectors a and b
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with 1 block and 256 threads per block
    vectorAdd<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy the result back from device to host
    hipMemcpy(c_i, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    

    // Free host memory
    delete[] a;
    delete[] b;
}

